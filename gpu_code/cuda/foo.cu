#include "hip/hip_runtime.h"
#include "hip/channel_descriptor.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "foo.cuh"
#include <thrust/device_vector.h>
#include <utility>
//#include  "kernel.cuh"
//#include <unistd.h>
#include <stdio.h>
//#include <windows.h>   // WinApi header
#include <stdio.h>
// for __syncthreads()
#ifndef __HIPCC__
#    define __HIPCC__
#endif

#include <hip/device_functions.h>

#ifdef __unix__
#    include <unistd.h>
#endif // __unix__
#include "progresscpp/ProgressBar.hpp"
extern int Min;
extern int Max;
extern float step;
extern int Size;
#define CHECK(res)                                                                       \
    {                                                                                    \
        if (res != hipSuccess)                                                          \
        {                                                                                \
            printf("Error ：%s:%d , ", __FILE__, __LINE__);                              \
            printf("code : %d , reason : %s \n", res, hipGetErrorString(res));          \
            exit(-1);                                                                    \
        }                                                                                \
    }

__device__ thrust::pair<float, float> find_ab(float *xpts, float *ypts, int i)
{
    float x1 = xpts[i * 2];
    float y1 = ypts[i * 2];
    float x2 = xpts[(i * 2) + 1];
    float y2 = ypts[(i * 2) + 1];

    //   printf("x1:%f y1:%f x2:%f y2:%f\n", x1, y1, x2, y2);
    float m = (y2 - y1) / (x2 - x1);
    float b = -m * x1 + y1;

    return thrust::make_pair(m, b);
}

__device__ bool isInside(float *xpts, float *ypts, int i, float x0, bool print = false)
{
    float x1 = xpts[i * 2];
    float x2 = xpts[(i * 2) + 1];
    if (print)
    {
        printf("s=%f xintersect=%f e=%f\n", x1, x0, x2);
    }

    if ((x1 > x2) && (x2 < x0) && (x0 < x1))
    {
        return true;
    }
    else if ((x1 < x2) && (x1 < x0) && (x0 < x2))
    {
        return true;
    }
    return false;
}

__global__ void findIntersection(float *xpts, float *ypts, int n, int batch, int start,count_t *d_mat,int Min,
                                 int Max,
                                 float step_,
                                 int Size)
{
    int i = threadIdx.x+start; // 0...batch size
    auto mb1 = find_ab(xpts, ypts, i);

    int step = (n/2-(i+1))/gridDim.x;
    int j = blockIdx.x*step+(i+1);
    int end = thrust::min(j+step,n/2);


    for (; j < end; j++)
    {
        auto mb2 = find_ab(xpts, ypts, j);
        double x_intersect = (mb2.second - mb1.second) / (mb1.first - mb2.first);
        if (isInside(xpts, ypts, j, x_intersect))
        {
            double yin = mb2.first * x_intersect + mb2.second;

            double x = 1/step_*(x_intersect+Max);
            double y = 1/step_*(yin+Max);
            int max_index = 2*Max*(int) (1/step_);
            if (y < 0 || y > max_index || x < 0 || x > max_index)
            {
                continue;
            }
            int xloc = __float2int_rn(x)*(float) max_index;
            int yloc = __float2int_rn(y);
            //            if(i==90 && j<200 && j>91){
            //                printf("%d %d %f %f x:%d y:%d\n",j+1,threadIdx.x + start+1,x_intersect,yin,__float2int_rn(x),yloc);
            //            }
            if(xloc+yloc>max_index*max_index)
                continue;
            atomicAdd(&d_mat[xloc+yloc],1.0f);

        }
    }
}

extern "C" float findValues(float *in_xpoints, float *in_ypoints, int num_pts,
    int batch_size, int start, float *xout, float *yout, count_t*mat)
{
    printf("%d\n",Min);
    float *d_pointsx;
    float *d_pointsy;

    float *d_xoutput;
    float *d_youtput;
    count_t * d_mat;

    int arraySize = sizeof(float) * num_pts;
    long int arraySize2 = sizeof(float) * num_pts * batch_size;

    //   printf("arraysize:%d\num_pts", arraySize2);

    CHECK(hipMalloc((void **)&d_pointsx, arraySize));
    CHECK(hipMalloc((void **)&d_pointsy, arraySize));
    CHECK(hipMalloc((void **)&d_mat, sizeof(count_t) * Size * Size));
    CHECK(hipMemcpy(d_mat, mat, sizeof(count_t) * Size * Size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_pointsx, in_xpoints, arraySize, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_pointsy, in_ypoints, arraySize, hipMemcpyHostToDevice));

    float time;
    hipEvent_t startt, stop;

    hipEventCreate(&startt);
    hipEventCreate(&stop);
    hipEventRecord(startt, 0);
    progresscpp::ProgressBar prog((num_pts / 2)/batch_size, 70);

    for (int s = 0; s < (num_pts / 2)-batch_size; s = s + batch_size)
    {
        findIntersection<<<10, batch_size>>>(d_pointsx, d_pointsy, num_pts,
            batch_size, s,d_mat,Min,Max,step,Size);

        prog.operator++();

        prog.display();
        CHECK(hipDeviceSynchronize());
    }
    prog.done();
    //    findIntersection<<<grid, block>>>(d_points, num_pts / 2, d_xoutput, d_youtput);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, startt, stop);
    CHECK(hipMemcpy(mat, d_mat, sizeof(count_t) * Size * Size, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_mat));
    CHECK(hipFree(d_pointsx));
    CHECK(hipFree(d_pointsy));
    return time;
}
